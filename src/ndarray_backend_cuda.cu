#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides
__device__ size_t convert_index(size_t index, CudaVec shape, CudaVec strides, size_t offset)
{
  size_t res = offset;
  for (int i = shape.size - 1; i >= 0; i--)
  {
    int j = index % shape.data[i];
    index /= shape.data[i];
    res += strides.data[i] * j;
  }
  return res;
}


__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size)
    out[gid] = a[convert_index(gid, shape, strides, offset)];
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}


__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset)
{
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size)
    out[convert_index(gid, shape, strides, offset)] = a[gid];
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
}


__global__ void ScalarSetitemKernel(scalar_t val, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset)
{
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size)
    out[convert_index(gid, shape, strides, offset)] = val;
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  CudaDims dim = CudaOneDim(size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                               VecToCuda(strides), offset);
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////


__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA arrays.
   * Args:
   *   a: Input array 'a' to be added
   *   b: Input array 'b' to be added
   *   out: Output array to store the result of 'a + b'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Kernel will execute on 'dim.grid' blocks, each containing 'dim.block' threads.
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

#define DEFINE_EWISE_OP_KERNEL(Op) \
  __global__ void Ewise##Op##Kernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
    if (gid < size) out[gid] = Op(a[gid], b[gid]); \
  }

#define DEFINE_EWISE_OP_FUNC(Op) \
  void Ewise##Op(const CudaArray& a, const CudaArray& b, CudaArray* out) { \
    CudaDims dim = CudaOneDim(out->size); \
    Ewise##Op##Kernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size); \
  }

#define DEFINE_EWISE_UNARY_OP_KERNEL(Op) \
  __global__ void Ewise##Op##Kernel(const scalar_t* a, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
    if (gid < size) out[gid] = Op(a[gid]); \
  }

#define DEFINE_EWISE_UNARY_OP_FUNC(Op) \
  void Ewise##Op(const CudaArray& a, CudaArray* out) { \
    CudaDims dim = CudaOneDim(out->size); \
    Ewise##Op##Kernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size); \
  }

#define DEFINE_SCALAR_OP_KERNEL(Op) \
  __global__ void Scalar##Op##Kernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) { \
    size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
    if (gid < size) out[gid] = Op(a[gid], val); \
  }

#define DEFINE_SCALAR_OP_FUNC(Op) \
  void Scalar##Op(const CudaArray& a, scalar_t val, CudaArray* out) { \
    CudaDims dim = CudaOneDim(out->size); \
    Scalar##Op##Kernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size); \
  }

__device__ scalar_t Mul(scalar_t a, scalar_t b) {
  return a * b;
}

__device__ scalar_t Div(scalar_t a, scalar_t b) {
  return a / b;
}

__device__ scalar_t Power(scalar_t a, scalar_t b) {
  return pow(a, b);
}

__device__ scalar_t Maximum(scalar_t a, scalar_t b) {
  return a > b ? a : b;
}

__device__ scalar_t Eq(scalar_t a, scalar_t b) {
  return a == b;
}

__device__ scalar_t Ge(scalar_t a, scalar_t b) {
  return a >= b;
}

__device__ scalar_t Log(scalar_t a) {
  return log(a);
}

__device__ scalar_t Exp(scalar_t a) {
  return exp(a);
}

__device__ scalar_t Tanh(scalar_t a) {
  return tanh(a);
}

DEFINE_EWISE_OP_KERNEL(Mul)
DEFINE_EWISE_OP_FUNC(Mul)
DEFINE_SCALAR_OP_KERNEL(Mul)
DEFINE_SCALAR_OP_FUNC(Mul)
DEFINE_EWISE_OP_KERNEL(Div)
DEFINE_EWISE_OP_FUNC(Div)
DEFINE_SCALAR_OP_KERNEL(Div)
DEFINE_SCALAR_OP_FUNC(Div)
DEFINE_SCALAR_OP_KERNEL(Power)
DEFINE_SCALAR_OP_FUNC(Power)
DEFINE_EWISE_OP_KERNEL(Maximum)
DEFINE_EWISE_OP_FUNC(Maximum)
DEFINE_SCALAR_OP_KERNEL(Maximum)
DEFINE_SCALAR_OP_FUNC(Maximum)
DEFINE_EWISE_OP_KERNEL(Eq)
DEFINE_EWISE_OP_FUNC(Eq)
DEFINE_SCALAR_OP_KERNEL(Eq)
DEFINE_SCALAR_OP_FUNC(Eq)
DEFINE_EWISE_OP_KERNEL(Ge)
DEFINE_EWISE_OP_FUNC(Ge)
DEFINE_SCALAR_OP_KERNEL(Ge)
DEFINE_SCALAR_OP_FUNC(Ge)
DEFINE_EWISE_UNARY_OP_KERNEL(Log)
DEFINE_EWISE_UNARY_OP_FUNC(Log)
DEFINE_EWISE_UNARY_OP_KERNEL(Exp)
DEFINE_EWISE_UNARY_OP_FUNC(Exp)
DEFINE_EWISE_UNARY_OP_KERNEL(Tanh)
DEFINE_EWISE_UNARY_OP_FUNC(Tanh)

////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication
////////////////////////////////////////////////////////////////////////////////

#define L 16
#define S 8
#define V 8

__device__ void cooperative_fetch_slice(const scalar_t *a, int a_m, int a_n, int offset_y, int offset_x, scalar_t *s, int s_m, int s_n)
{
  int nthreads = blockDim.y * blockDim.x;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  for (int i = 0; i < s_m * s_n; i += nthreads)
  {
    int j = i + tid;
    if (j >= s_m * s_n)
      break;
    int y = j / s_n;
    int x = j % s_n;
    if (y + offset_y >= a_m || x + offset_x >= a_n)
      s[y*s_n+x] = 0;
    else
      s[y*s_n+x] = a[convert_index(j, {2, {s_m, s_n}}, {2, {a_n, 1}}, offset_y * a_n + offset_x)];
  }
}

__global__ void MatmulKernel(const scalar_t *A, const scalar_t *B, scalar_t *out, uint32_t M, uint32_t N,
            uint32_t P)
{
  __shared__ scalar_t sA[L][S], sB[S][L];
  scalar_t c[V][V] = {0};
  scalar_t a[V], b[V];

  for (int ko = 0; ko < N; ko += S)
  {
    /* Cooperative Fetching */
    // sA
    cooperative_fetch_slice(A, M, N, blockIdx.y * L, ko, sA[0], L, S);
    // sB
    cooperative_fetch_slice(B, N, P, ko, blockIdx.x * L, sB[0], S, L);
    __syncthreads();

    for (int ki = 0; ki < S; ki++)
    {
      for (int i = 0; i < V; i++)
      {
        a[i] = sA[threadIdx.y * V + i][ki];
        b[i] = sB[ki][threadIdx.x * V + i];
      }
      for (int y = 0; y < V; y++)
      {
        for (int x = 0; x < V; x++)
        {
          c[y][x] += a[y] * b[x];
        }
      }
    }
  }

  int ybase = (blockIdx.y * blockDim.y + threadIdx.y) * V;
  int xbase = (blockIdx.x * blockDim.x + threadIdx.x) * V;
  for (int y = 0; y < V; y++)
  {
    for (int x = 0; x < V; x++)
    {
      if (ybase + y >= M || xbase + x >= P)
      {
        continue;
      }
      out[(ybase+y)*P+xbase+x] = c[y][x];
    }
  }

}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  CudaDims dim;
  dim.block = dim3(L/V, L/V);
  dim.grid = dim3((P+L-1)/L, (M+L-1)/L);
  MatmulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size)
  {
    int offset = gid * reduce_size;
    out[gid] = a[offset];
    for (int i = offset + 1; i < offset + reduce_size; i++)
    {
      if (a[i] > out[gid])
      {
        out[gid] = a[i];
      }
    }
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
}


__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t size, size_t reduce_size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size)
  {
    int offset = gid * reduce_size;
    out[gid] = 0;
    for (int i = offset; i < offset + reduce_size; i++)
    {
      out[gid] += a[i];
    }
  }
}

void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, reduce_size);
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
